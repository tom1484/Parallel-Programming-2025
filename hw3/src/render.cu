#include "hip/hip_runtime.h"
#include <cstring>
#include <iostream>

#include "arithmic.hpp"
#include "render.hpp"
#include "schedule.hpp"
#include "utils.hpp"

extern uint width;        // image width
extern uint height;       // image height
extern vec2 iResolution;  // just for convenience of calculation

extern vec3 camera_pos;  // camera position in 3D space (x, y, z)
extern vec3 target_pos;  // target position in 3D space (x, y, z)

extern ScheduleDim dim;

__constant__ uint d_width;
__constant__ uint d_height;
__constant__ vec2 d_iResolution;
__constant__ vec3 d_camera_pos;
__constant__ vec3 d_target_pos;
__constant__ ScheduleDim d_dim;

void copy_constants_to_device() {
    hipMemcpyToSymbol(HIP_SYMBOL(d_width), &width, sizeof(uint));
    hipMemcpyToSymbol(HIP_SYMBOL(d_height), &height, sizeof(uint));
    hipMemcpyToSymbol(HIP_SYMBOL(d_iResolution), &iResolution, sizeof(vec2));
    hipMemcpyToSymbol(HIP_SYMBOL(d_camera_pos), &camera_pos, sizeof(vec3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_target_pos), &target_pos, sizeof(vec3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dim), &dim, sizeof(ScheduleDim));
}

__device__ float _estimate(vec3 pos, float& trap) {
    vec3 v = pos;
    float dr = 1.f;         // |v'|
    float r = __length(v);  // r = |v| = sqrt(x^2 + y^2 + z^2)
    trap = r;

    for (int i = 0; i < md_iter; ++i) {
        float theta = atan2f(v.y, v.x) * power;
        float phi = asinf(__div(v.z, r)) * power;

        float sin_theta, cos_theta;
        float sin_phi, cos_phi;
        __sincosf(theta, &sin_theta, &cos_theta);
        __sincosf(phi, &sin_phi, &cos_phi);

        float r_pow, r_pow_less;
        __mendel_pow(r, r_pow_less, r_pow);

        // update vk+1
        v = __fma(vec3(cos_theta * cos_phi, cos_phi * sin_theta, -sin_phi), r_pow, pos);
        // update dr
        dr = __fma(power * r_pow_less, dr, 1.f);
        // orbit trap for coloring
        trap = __min(trap, r);

        r = __length(v);         // update r
        if (r > bailout) break;  // if escaped
    }

    return 0.5f * logf(r) * __div(r, dr);  // mandelbulb's DE function
}

__device__ float _map_trap(vec3 pos, float& trap) {
    // rotation matrix, rotate 90 deg (pi/2) along the X-axis
    // vec2 rt = vec2(0.f, 1.f);
    // vec3 rp = mat3(1.f, 0.f, 0.f, 0.f, rt.x, -rt.y, 0.f, rt.y, rt.x) * pos;
    vec3 rp = vec3(pos.x, -pos.z, pos.y);
    return _estimate(rp, trap);
}

__device__ float _map(vec3 pos) {
    float _trap;  // dummy
    return _map_trap(pos, _trap);
}

__device__ vec3 _palette(float t, vec3 a, vec3 b, vec3 c, vec3 d) {
    return __fma(__cos(2.f * PI * __fma(c, t, d)), b, a);
}

__device__ float _softshadow(vec3 origin, vec3 direction, float k) {
    float res = 1.0f;
    float t = 0.f;  // total distance
    for (int i = 0; i < shadow_step; ++i) {
        float h = _map(__fma(direction, t, origin));
        // closer to the objects, k*h/t terms will produce darker shadow
        res = __min(res, k * __div(h, t));
        if (res < 0.02f) return 0.02f;
        t += __clamp(h, .001f, step_limiter);  // move ray
    }
    return __clamp(res, .02f, 1.f);
}

__device__ float _trace_ray(vec3 origin, vec3 direction, float& trap) {
    float total_dis = 0;  // total distance
    float len = 0;        // current distance

    for (int i = 0; i < ray_step; ++i) {
        // get minimum distance from current ray position to the object's surface
        len = _map_trap(__fma(direction, total_dis, origin), trap);
        if (__abs(len) < eps || total_dis > far_plane) break;
        total_dis = __fma(len, ray_multiplier, total_dis);
    }
    // If exceeds the far plane then return -1 which means the ray missed a shot
    return total_dis < far_plane ? total_dis : -1.f;
}

// use gradient to calc surface normal
__device__ vec3 calculate_norm(vec3 p) {
    vec2 e = vec2(eps, 0.f);
    return __normalize(vec3(_map(p + e.xyy()) - _map(p - e.xyy()),  // dx
                            _map(p + e.yxy()) - _map(p - e.yxy()),  // dy
                            _map(p + e.yyx()) - _map(p - e.yyx())   // dz
                            ));
}

__global__ void __launch_bounds__(256, 4) _render_pixel(uchar* buffer) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    // Round-robin scheduling
    // int x = d_dim.n_blocks_x * threadIdx.x + blockIdx.x;
    // int y = d_dim.n_blocks_y * threadIdx.y + blockIdx.y;
    if (x >= d_width || y >= d_height) return;

    int pixel_index = (y * d_width + x) * 4;

    float final_color_r = 0.0f;
    float final_color_g = 0.0f;
    float final_color_b = 0.0f;

    // Create camera
    vec3 origin = d_camera_pos;                                      // ray (camera) origin
    vec3 target = d_target_pos;                                      // target position
    vec3 forward = __normalize(target - origin);                     // forward vector
    vec3 side = __normalize(__cross(forward, vec3(0.f, 1.f, 0.f)));  // right (side) vector
    vec3 up = __normalize(__cross(side, forward));                   // up vector

    vec2 uv_pos = vec2(x << 1, y << 1) - d_iResolution.xy();

    for (int m = 0; m < AA; ++m) {
        for (int n = 0; n < AA; ++n) {
            // Convert screen space coordinate to (-ap~ap, -1~1)
            vec2 uv = (__fma(vec2(m, n), __ric(HALF_AA), uv_pos)) / d_iResolution.y;
            uv.y *= -1;  // flip upside down
            // ray direction
            vec3 direction = __normalize(uv.x * side + uv.y * up + FOV * forward);

            float trap;
            float depth = _trace_ray(origin, direction, trap);

            // Lighting
            vec3 color(0.f);                             // color
            vec3 light_dir = __normalize(d_camera_pos);  // sun direction (directional light)
            vec3 light_color = vec3(1.f, .9f, .717f);    // light color

            // Coloring
            if (depth < 0.f) {      // miss (hit sky)
                color = vec3(0.f);  // sky color (black)
            } else {
                vec3 pos = origin + direction * depth;          // hit position
                vec3 nr = calculate_norm(pos);                  // get surface normal
                vec3 hal = __normalize(light_dir - direction);  // blinn-phong lighting model (vector h)

                // use orbit trap to get the color
                color = _palette(trap - .4f, vec3(.5f), vec3(.5f), vec3(1.f), vec3(.0f, .1f, .2f));  // diffuse color
                vec3 ambient_color = vec3(0.3f);                                                     // ambient color
                float gloss = 32.f;                                                                  // specular gloss

                // simple blinn phong lighting model
                float ambient = __fma(0.3f, nr.y, 0.7f) *
                                __fma(0.8f, __clamp(0.05f * (float)logf(trap), 0.0f, 1.0f), 0.2f);  // self occlution
                float shadow = _softshadow(__fma(nr, .001f, pos), light_dir, 16.f);                 // shadow
                float diffuse = __clamp(__dot(light_dir, nr), 0.f, 1.f) * shadow;                   // diffuse
                float specular = __pow(__clamp(__dot(nr, hal), 0.f, 1.f), gloss) * diffuse;         // self shadow

                vec3 lin = ambient_color * __fma(.95f, ambient, .05f);
                lin = __fma(light_color, diffuse * 0.8f, lin);  // diffuse * light color * light intensity
                color *= lin;

                color = __pow(color, vec3(.7f, .9f, 1.f));  // fake SSS (subsurface scattering)
                color += specular * 0.8f;                   // specular
            }

            color = __clamp(__pow(color, vec3(.4545f)), 0.f, 1.f);  // gamma correction
            // fcol += vec4(col, 1.f);
            final_color_r += color.r;
            final_color_g += color.g;
            final_color_b += color.b;
        }
    }
    // convert float (0~1) to unsigned char (0~255)
    // fcol /= (float)(AA * AA);
    // fcol *= 255.0f;
    float scaling = __div(255.0f, (float)(AA * AA));
    uchar color_r = final_color_r * scaling;
    uchar color_g = final_color_g * scaling;
    uchar color_b = final_color_b * scaling;

    *(uchar4*)(buffer + pixel_index) = make_uchar4(color_r, color_g, color_b, 255);
}

void render(uchar* raw_image) {
    uchar* d_buffer;
    hipMalloc((void**)&d_buffer, width * height * 4);
    copy_constants_to_device();

    // Set dimensions
    dim3 gridDim(dim.n_blocks_x, dim.n_blocks_y);
    dim3 blockDim(dim.n_threads_x, dim.n_threads_y);

#ifdef DEBUG
    int block_size = dim.n_threads_x * dim.n_threads_y;
    estimate_occupancy((void*)_render_pixel, block_size, 0);
#endif

    _render_pixel<<<gridDim, blockDim>>>(d_buffer);
    CUDA_CHECK(hipDeviceSynchronize());

    hipMemcpy(raw_image, d_buffer, width * height * 4, hipMemcpyDeviceToHost);
    hipFree(d_buffer);
}
